#include "sixtracklib/cuda/cuda_env.h"

#include <stddef.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "sixtracklib/_impl/definitions.h"
#include "sixtracklib/common/blocks.h"
#include "sixtracklib/cuda/impl/track_particles_kernel.cuh"
#include "sixtracklib/common/pyheadtail_particles.h"

extern __host__ bool NS(Track_particles_on_cuda_gpu_part)(
    int const num_of_blocks,
    int const num_threads_per_block,
    SIXTRL_UINT64_T const num_of_turns,
    NS(Blocks)* SIXTRL_RESTRICT particles_buffer,
    NS(Blocks)* SIXTRL_RESTRICT beam_elements,
    NS(Blocks)* SIXTRL_RESTRICT elem_by_elem_buffer,
    ParticleData *p
    );

bool __host__ NS(Track_particles_on_cuda_gpu_part)(
    int const num_of_blocks,
    int const num_threads_per_block,
    SIXTRL_UINT64_T const num_of_turns,
    NS(Blocks)* SIXTRL_RESTRICT particles_buffer,
    NS(Blocks)* SIXTRL_RESTRICT beam_elements,
    NS(Blocks)* SIXTRL_RESTRICT elem_by_elem_buffer,
    ParticleData *p
    )
{
    double *x = p->x;
    double *xp = p->xp;
    double *y = p->y;
    double *yp = p->yp;
    double *q0 = p->q0;
    double *mass0 = p->mass0;
    double *beta0 = p->beta0;
    double *gamma0 = p->gamma0;
    double *z = p->z;
    double *dp = p->dp;
    double *p0c = p->p0c;
    bool success = false;

    if( ( NS(Blocks_are_serialized)( particles_buffer ) ) &&
        ( NS(Blocks_are_serialized)( beam_elements ) ) )
    {
        hipError_t err;

        SIXTRL_STATIC_VAR uint64_t const U64_ZERO =
            static_cast< uint64_t >( 0 );

        uint64_t dummy_elem_by_elem_header[ 4 ] =
        {
            U64_ZERO, U64_ZERO, U64_ZERO ,U64_ZERO
        };

        /* ----------------------------------------------------------------- */

        unsigned char* cuda_particles_data_buffer = NULL;
        unsigned char* host_particles_data_buffer = NULL;

        NS(block_size_t) const particles_buffer_size =
            NS(Blocks_get_total_num_bytes)( particles_buffer );

        if( particles_buffer_size > 0u )
        {
            err = hipMalloc( ( void** )&cuda_particles_data_buffer,
                              particles_buffer_size );

            success  = ( err == hipSuccess );
            success &= ( cuda_particles_data_buffer != NULL );

            host_particles_data_buffer = NS(Blocks_get_data_begin)(
                particles_buffer );

            err = hipMemcpy(
                cuda_particles_data_buffer, host_particles_data_buffer,
                particles_buffer_size, hipMemcpyHostToDevice );

            success &= ( err == hipSuccess );
        }

        /* ----------------------------------------------------------------- */

        unsigned char* cuda_beam_elements_data_buffer = NULL;
        unsigned char* host_beam_elements_data_buffer = NULL;

        NS(block_size_t) const beam_elements_buffer_size =
            NS(Blocks_get_total_num_bytes)( beam_elements );

        if( success )
        {
            if( beam_elements_buffer_size > 0u )
            {
                err = hipMalloc( ( void** )&cuda_beam_elements_data_buffer,
                                    beam_elements_buffer_size );

                success  = ( err == hipSuccess );
                success &= ( cuda_beam_elements_data_buffer != NULL );

                host_beam_elements_data_buffer = NS(Blocks_get_data_begin)(
                    beam_elements );

                err = hipMemcpy( cuda_beam_elements_data_buffer,
                    host_beam_elements_data_buffer, beam_elements_buffer_size,
                    hipMemcpyHostToDevice );

                success &= ( err == hipSuccess );
            }
            else
            {
                success = false;
            }
        }

        /* ----------------------------------------------------------------- */

        bool use_elem_by_elem_buffer = false;
        unsigned char* cuda_elem_by_elem_data_buffer = NULL;
        unsigned char* host_elem_by_elem_data_buffer = NULL;

        NS(block_size_t) elem_by_elem_buffer_size =
            ( NS(Blocks_are_serialized)( elem_by_elem_buffer ) )
                ? NS(Blocks_get_total_num_bytes)( elem_by_elem_buffer ) : 0u;

        if( success )
        {
            if( elem_by_elem_buffer_size > 0u )
            {
                use_elem_by_elem_buffer = true;
                host_elem_by_elem_data_buffer = NS(Blocks_get_data_begin)(
                    elem_by_elem_buffer );

                err = hipMalloc( ( void** )&cuda_elem_by_elem_data_buffer,
                                  elem_by_elem_buffer_size );
            }
            else
            {
                host_elem_by_elem_data_buffer =
                    ( unsigned char* )&dummy_elem_by_elem_header[ 0 ];

                elem_by_elem_buffer_size = 4u * sizeof( uint64_t );

                err = hipMalloc( ( void** )&cuda_elem_by_elem_data_buffer,
                                  4u * sizeof( uint64_t ) );
            }

            success  = ( err == hipSuccess );
            success &= ( cuda_elem_by_elem_data_buffer != NULL );
            success &= ( host_elem_by_elem_data_buffer != NULL );
            success &= ( elem_by_elem_buffer_size > 0u );

            use_elem_by_elem_buffer &= success;
        }

        if( success )
        {
            err = hipMemcpy(
                cuda_elem_by_elem_data_buffer,
                host_elem_by_elem_data_buffer,
                elem_by_elem_buffer_size, hipMemcpyHostToDevice );

            success = ( err == hipSuccess );
        }

        int64_t* cuda_success_flag = NULL;
        int64_t  host_success_flag = static_cast< int64_t >( 0 );

        if( success )
        {
            err = hipMalloc( ( void** )&cuda_success_flag,
                                sizeof( int64_t ) );

            success = ( err == hipSuccess );
        }

        if( success )
        {
            err = hipMemcpy( cuda_success_flag, &host_success_flag,
                              sizeof( int64_t ), hipMemcpyHostToDevice );

            success = ( err == hipSuccess );
        }

        if( success )
        {
            Track_remap_serialized_blocks_buffer<<<
                num_of_blocks, num_threads_per_block >>>(
                    cuda_particles_data_buffer, cuda_beam_elements_data_buffer,
                    cuda_elem_by_elem_data_buffer, cuda_success_flag );

            err = hipDeviceSynchronize();
            success  = ( err == hipSuccess );

            if( success )
            {
                err = hipMemcpy( &host_success_flag, cuda_success_flag,
                                  sizeof( int64_t ), hipMemcpyDeviceToHost );
                success  = ( err == hipSuccess );
                success &= ( host_success_flag == static_cast<int64_t>( 0 ) );
            }
        }

	if ( success )
	{
            Copy_buffer_pyheadtail_sixtracklib<<<
                num_of_blocks, num_threads_per_block >>>(
                    cuda_particles_data_buffer, 
                    x, xp, y, yp, q0, mass0, beta0, gamma0, z, dp, p0c, 
		    cuda_success_flag );

            err = hipDeviceSynchronize();
            success  = ( err == hipSuccess );

	}

        if( success )
        {
            Track_particles_kernel_cuda<<< num_of_blocks, num_threads_per_block >>>(
                num_of_turns, cuda_particles_data_buffer,
                cuda_beam_elements_data_buffer, cuda_elem_by_elem_data_buffer,
                cuda_success_flag );

            err = hipDeviceSynchronize();
            success = ( err == hipSuccess );
	    if ( success )
	    {
		    Copy_buffer_sixtracklib_pyheadtail<<<
		    num_of_blocks, num_threads_per_block >>>(
                    cuda_particles_data_buffer,
                    x, xp, y, yp, q0, mass0, beta0, gamma0, z, dp, p0c,
		     cuda_success_flag );
            	    err = hipDeviceSynchronize();
            	    success = ( err == hipSuccess );
	    }
        }

        err = hipFree( cuda_success_flag );
        success &= ( err == hipSuccess );

        err = hipFree( cuda_elem_by_elem_data_buffer  );
        success &= ( err == hipSuccess );

        err = hipFree( cuda_beam_elements_data_buffer );
        success &= ( err == hipSuccess );

        err = hipFree( cuda_particles_data_buffer  );
        success &= ( err == hipSuccess );

    }

    return success;
}

extern __host__ bool NS(Track_particles_on_cuda)(
    int const num_of_blocks,
    int const num_threads_per_block,
    SIXTRL_UINT64_T const num_of_turns,
    NS(Blocks)* SIXTRL_RESTRICT particles_buffer,
    NS(Blocks)* SIXTRL_RESTRICT beam_elements,
    NS(Blocks)* SIXTRL_RESTRICT elem_by_elem_buffer );

bool __host__ NS(Track_particles_on_cuda)(
    int const num_of_blocks,
    int const num_threads_per_block,
    SIXTRL_UINT64_T const num_of_turns,
    NS(Blocks)* SIXTRL_RESTRICT particles_buffer,
    NS(Blocks)* SIXTRL_RESTRICT beam_elements,
    NS(Blocks)* SIXTRL_RESTRICT elem_by_elem_buffer )
{
    bool success = false;

    if( ( NS(Blocks_are_serialized)( particles_buffer ) ) &&
        ( NS(Blocks_are_serialized)( beam_elements ) ) )
    {
        hipError_t err;

        SIXTRL_STATIC_VAR uint64_t const U64_ZERO =
            static_cast< uint64_t >( 0 );

        uint64_t dummy_elem_by_elem_header[ 4 ] =
        {
            U64_ZERO, U64_ZERO, U64_ZERO ,U64_ZERO
        };

        /* ----------------------------------------------------------------- */

        unsigned char* cuda_particles_data_buffer = NULL;
        unsigned char* host_particles_data_buffer = NULL;

        NS(block_size_t) const particles_buffer_size =
            NS(Blocks_get_total_num_bytes)( particles_buffer );

        if( particles_buffer_size > 0u )
        {
            err = hipMalloc( ( void** )&cuda_particles_data_buffer,
                              particles_buffer_size );

            success  = ( err == hipSuccess );
            success &= ( cuda_particles_data_buffer != NULL );

            host_particles_data_buffer = NS(Blocks_get_data_begin)(
                particles_buffer );

            err = hipMemcpy(
                cuda_particles_data_buffer, host_particles_data_buffer,
                particles_buffer_size, hipMemcpyHostToDevice );

            success &= ( err == hipSuccess );
        }

        /* ----------------------------------------------------------------- */

        unsigned char* cuda_beam_elements_data_buffer = NULL;
        unsigned char* host_beam_elements_data_buffer = NULL;

        NS(block_size_t) const beam_elements_buffer_size =
            NS(Blocks_get_total_num_bytes)( beam_elements );

        if( success )
        {
            if( beam_elements_buffer_size > 0u )
            {
                err = hipMalloc( ( void** )&cuda_beam_elements_data_buffer,
                                    beam_elements_buffer_size );

                success  = ( err == hipSuccess );
                success &= ( cuda_beam_elements_data_buffer != NULL );

                host_beam_elements_data_buffer = NS(Blocks_get_data_begin)(
                    beam_elements );

                err = hipMemcpy( cuda_beam_elements_data_buffer,
                    host_beam_elements_data_buffer, beam_elements_buffer_size,
                    hipMemcpyHostToDevice );

                success &= ( err == hipSuccess );
            }
            else
            {
                success = false;
            }
        }

        /* ----------------------------------------------------------------- */

        bool use_elem_by_elem_buffer = false;
        unsigned char* cuda_elem_by_elem_data_buffer = NULL;
        unsigned char* host_elem_by_elem_data_buffer = NULL;

        NS(block_size_t) elem_by_elem_buffer_size =
            ( NS(Blocks_are_serialized)( elem_by_elem_buffer ) )
                ? NS(Blocks_get_total_num_bytes)( elem_by_elem_buffer ) : 0u;

        if( success )
        {
            if( elem_by_elem_buffer_size > 0u )
            {
                use_elem_by_elem_buffer = true;
                host_elem_by_elem_data_buffer = NS(Blocks_get_data_begin)(
                    elem_by_elem_buffer );

                err = hipMalloc( ( void** )&cuda_elem_by_elem_data_buffer,
                                  elem_by_elem_buffer_size );
            }
            else
            {
                host_elem_by_elem_data_buffer =
                    ( unsigned char* )&dummy_elem_by_elem_header[ 0 ];

                elem_by_elem_buffer_size = 4u * sizeof( uint64_t );

                err = hipMalloc( ( void** )&cuda_elem_by_elem_data_buffer,
                                  4u * sizeof( uint64_t ) );
            }

            success  = ( err == hipSuccess );
            success &= ( cuda_elem_by_elem_data_buffer != NULL );
            success &= ( host_elem_by_elem_data_buffer != NULL );
            success &= ( elem_by_elem_buffer_size > 0u );

            use_elem_by_elem_buffer &= success;
        }

        if( success )
        {
            err = hipMemcpy(
                cuda_elem_by_elem_data_buffer,
                host_elem_by_elem_data_buffer,
                elem_by_elem_buffer_size, hipMemcpyHostToDevice );

            success = ( err == hipSuccess );
        }

        int64_t* cuda_success_flag = NULL;
        int64_t  host_success_flag = static_cast< int64_t >( 0 );

        if( success )
        {
            err = hipMalloc( ( void** )&cuda_success_flag,
                                sizeof( int64_t ) );

            success = ( err == hipSuccess );
        }

        if( success )
        {
            err = hipMemcpy( cuda_success_flag, &host_success_flag,
                              sizeof( int64_t ), hipMemcpyHostToDevice );

            success = ( err == hipSuccess );
        }

        if( success )
        {
            Track_remap_serialized_blocks_buffer<<<
                num_of_blocks, num_threads_per_block >>>(
                    cuda_particles_data_buffer, cuda_beam_elements_data_buffer,
                    cuda_elem_by_elem_data_buffer, cuda_success_flag );

            err = hipDeviceSynchronize();
            success  = ( err == hipSuccess );

            if( success )
            {
                err = hipMemcpy( &host_success_flag, cuda_success_flag,
                                  sizeof( int64_t ), hipMemcpyDeviceToHost );

                success  = ( err == hipSuccess );
                success &= ( host_success_flag == static_cast<int64_t>( 0 ) );
            }
        }

        if( success )
        {
            Track_particles_kernel_cuda<<< num_of_blocks, num_threads_per_block >>>(
                num_of_turns, cuda_particles_data_buffer,
                cuda_beam_elements_data_buffer, cuda_elem_by_elem_data_buffer,
                cuda_success_flag );

            err = hipDeviceSynchronize();
            success = ( err == hipSuccess );

            if( success )
            {
                err = hipMemcpy( &host_success_flag, cuda_success_flag,
                                  sizeof( int64_t ), hipMemcpyDeviceToHost );

                success  = ( err == hipSuccess );
                success &= ( host_success_flag == U64_ZERO );
            }

            if( success )
            {
                err = hipMemcpy( host_particles_data_buffer,
                                  cuda_particles_data_buffer,
                                  particles_buffer_size,
                                  hipMemcpyDeviceToHost );

                success = ( err == hipSuccess );
            }

            if( ( success ) && ( use_elem_by_elem_buffer ) )
            {
                err = hipMemcpy( host_elem_by_elem_data_buffer,
                                  cuda_elem_by_elem_data_buffer,
                                  elem_by_elem_buffer_size,
                                  hipMemcpyDeviceToHost );

                success = ( err == hipSuccess );
            }
        }

        if( success )
        {
            success = ( 0 == NS(Blocks_unserialize)(
                particles_buffer, host_particles_data_buffer ) );
        }

        if( ( success ) && ( use_elem_by_elem_buffer ) )
        {
            success &= ( 0 == NS(Blocks_unserialize)(
                elem_by_elem_buffer, host_elem_by_elem_data_buffer ) );
        }

        err = hipFree( cuda_success_flag );
        success &= ( err == hipSuccess );

        err = hipFree( cuda_elem_by_elem_data_buffer  );
        success &= ( err == hipSuccess );

        err = hipFree( cuda_beam_elements_data_buffer );
        success &= ( err == hipSuccess );

        err = hipFree( cuda_particles_data_buffer  );
        success &= ( err == hipSuccess );
    }

    return success;
}

